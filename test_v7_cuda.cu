#include "hip/hip_runtime.h"
#include <limits>

#include "soa_v7_cuda.h"

#include <hiprand/hiprand_kernel.h>

#include <Eigen/Core>
#include <Eigen/Geometry>

#define CUDA_UNIT_CHECK(A) CPPUNIT_ASSERT_EQUAL(hipSuccess, A)

namespace {
  // fill element
  template <class T>
  __host__ __device__ __forceinline__ void fillElement(T & e, size_t i) {
    e.x = 11.0 * i;
    e.y = 22.0 * i;
    e.z = 33.0 * i;
    e.colour = i;
    e.value = 0x10001 * i;
    e.py = &e.y;
  }

  // Fill up the elements of the SoA
  __global__ void fillSoA(testSoA::SoA soa) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= soa.nElements()) return;
    // compiler does not belive we can use a temporary soa[i] to store results.
    // So make an lvalue.
    auto e = soa[i];
    fillElement(e, i);
  }
  
  // Fill elements with random data.
  __global__ void randomFillSoA(testSoA::SoA soa, uint64_t seed) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= soa.nElements()) return;
    hiprandState state;
    hiprand_init(seed + i, 0, 0, &state);
    soa[i].x = hiprand_uniform_double(&state);
    soa[i].y = hiprand_uniform_double(&state);
    soa[i].z = hiprand_uniform_double(&state);
  }

  __global__ void fillAoS(testSoA::AoSelement *aos, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    fillElement(aos[i], i);
  }

  // Simple cross product for elements
  template <typename T, typename T2>
  __host__ __device__ __forceinline__ void crossProduct(T & r, const T2 & a, const T2 & b) {
    r.x = a.y * b.z - a.z * b.y;
    r.y = a.z * b.x - a.x * b.z;
    r.z = a.x * b.y - a.y * b.x;
  }

  // Simple indiredt cross product (SoA)
  __global__ void indirectCrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    // C++ does not allow creating non-const references to temporary variables
    // this workaround makes the temporary variable 
    auto ri = r[i];
    crossProduct(ri, a[i], b[i]);
  }

  // Simple direct cross product (SoA)
  __global__ void directCrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    r[i].x = a[i].y * b[i].z - a[i].z * b[i].y;
    r[i].y = a[i].z * b[i].x - a[i].x * b[i].z;
    r[i].z = a[i].x * b[i].y - a[i].y * b[i].x;
  }

  // Hand-made cross product as a reference (SoA)
  __global__ void handcraftedCrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    r.x()[i] = a.y()[i] * b.y()[i] - a.z()[i] * b.y()[i];
    r.y()[i] = a.z()[i] * b.x()[i] - a.x()[i] * b.z()[i];
    r.z()[i] = a.x()[i] * b.y()[i] - a.y()[i] * b.x()[i];
  }
  
  using V3 = Eigen::Vector3d;
  using DynStride = Eigen::InnerStride<Eigen::Dynamic>;
  using CStride = Eigen::InnerStride<1024>;
  using MapV3 =  Eigen::Map<V3,0, DynStride>;
  using CMapV3 =  Eigen::Map<const V3,0,  DynStride>;
  
   // Eigen based cross product
  __global__ void eigenCrossProductSoA(double* rx, const double* ax, const double* bx, size_t nElements, size_t stride) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;

    CMapV3 ma(ax+i, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(stride));
    CMapV3 mb(bx+i, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(stride));
    MapV3 mr(rx+i, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(stride));
    mr = ma.cross(mb);
  }

  // Simple cross product (SoA on CPU)
  __host__ void indirectCPUcrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    for (size_t i =0; i< nElements; ++i) {
      // This version is also affected.
      auto ri = r[i];
      crossProduct(ri, a[i], b[i]);
    }
  }

  // Simple cross product (AoS)
  __global__ void crossProductAoS(testSoA::AoSelement *r,
          testSoA::AoSelement *a, testSoA::AoSelement *b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    crossProduct(r[i], a[i], b[i]);
  }

  void hexdump(void *ptr, int buflen) {
    /* From https://stackoverflow.com/a/29865 with adaptations */
    unsigned char *buf = (unsigned char*)ptr;
    int i, j;
    for (i=0; i<buflen; i+=16) {
      printf("%06x: ", i);
      for (j=0; j<16; j++) {
        if (i+j < buflen)
          printf("%02x ", buf[i+j]);
        else
          printf("   ");
        if ((i+j) % 4 == 3) printf (" ");
      }
      printf(" ");
  //  for (j=0; j<16; j++)
  //    if (i+j < buflen)
  //      printf("%c", isprint(buf[i+j]) ? buf[i+j] : '.');
      printf("\n");
    }
  }
  // Check we find what we wanted to initialize.
  // Pass should be initialized to true.
  template <class T>
  __host__ __device__ __forceinline__ void checkSoAelement(T soa, size_t i, bool & pass) {
    if (i >= soa.nElements() || !pass) return;
    if (soa[i].x != 11.0 * i) { pass = false; return; }
    if (soa[i].y != 22.0 * i) { pass = false; return; }
    if (soa[i].z != 33.0 * i) { pass = false; return; }
    if (soa[i].colour != i) { pass = false; return; }
    if (soa[i].value != static_cast<int32_t>(0x10001 * i)) { pass = false; return; }
  }

  // Check r[i].{x, y, z} are close enough to zero compared to a[i].{x,y,z} and b[i].{x,y,z}
  // to validate a cross product of a vector with itself produced a zero (enough) result.
  template <class T>
  __host__ __device__ __forceinline__ void checkSoAelementNullityRealtiveToSquare(T resSoA, T referenceSoA, size_t i, double epsilon, bool & pass) {
    if (i >= resSoA.nElements() || !pass) return;
    auto ref = max (abs(referenceSoA[i].x), 
                    max(abs(referenceSoA[i].y), 
                        abs(referenceSoA[i].z)));
    ref *= ref * epsilon;
    if (abs(resSoA[i].x) > ref) { pass = false; return; }
    if (abs(resSoA[i].y) > ref) { pass = false; return; }
    if (abs(resSoA[i].z) > ref) { pass = false; return; }    
  }

  // Check r[i].{x, y, z} are close enough to zero compared to a[i].{x,y,z} and b[i].{x,y,z}
  // to validate a cross product of a vector with itself produced a zero (enough) result.
  template <class T>
  __host__ __device__ __forceinline__ void checkCrossProduct(T resultSoA, T aSoA, T bSoA, size_t i, double epsilon, bool & pass) {
    if (i >= resultSoA.nElements() || !pass) return;
    auto refA = max (abs(aSoA[i].x), 
                     max(abs(aSoA[i].y), 
                         abs(aSoA[i].z)));
    auto refB = max (abs(bSoA[i].x), 
                     max(abs(bSoA[i].y), 
                         abs(bSoA[i].z)));
    auto ref = max(refA, refB);
    ref *= ref * epsilon;
    testSoA::AoSelement myRes;
    crossProduct(myRes, aSoA[i], bSoA[i]);
    if (abs(myRes.x - resultSoA[i].x) > ref) { pass = false; return; }
    if (abs(myRes.y - resultSoA[i].y) > ref) { pass = false; return; }
    if (abs(myRes.z - resultSoA[i].z) > ref) { pass = false; return; }
  }
}

void testSoA::fill() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  CUDA_UNIT_CHECK(hipSetDevice(defaultDevice));  
  hipStream_t stream;
  auto e = hipStreamCreate(&stream);
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors
  auto deviceSoABlock = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlock = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoA(deviceSoABlock.get(), elementsCount);
  SoA hostSoA(hostSoABlock.get(), elementsCount);
  
  // Call kernel, get result
  fillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoA);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlock.get(), deviceSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoA.nElements(); i++) checkSoAelement(hostSoA, i, pass);
  if (!pass) {
    std::cout << "In " << typeid(*this).name() << " check failed at i= " << i << ")" << std::endl;
    hexdump(hostSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()));
    printf("base=%p, &y=%p\n", deviceSoABlock.get(), deviceSoA.y());
  }
  CPPUNIT_ASSERT(pass);
}

void testSoA::randomFill() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  CUDA_UNIT_CHECK(hipSetDevice(defaultDevice));  
  hipStream_t stream;
  auto e = hipStreamCreate(&stream);
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors
  auto deviceSoABlock = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlock = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoA(deviceSoABlock.get(), elementsCount);
  SoA hostSoA(hostSoABlock.get(), elementsCount);
  
  // Call kernel, get result
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoA, 0xbaddeed5);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlock.get(), deviceSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));
}

void testSoA::crossProduct() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t stream;
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlockA = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockR = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockA = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockR = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoAA(deviceSoABlockA.get(), elementsCount);
  SoA deviceSoAR(deviceSoABlockR.get(), elementsCount);
  SoA hostSoAA(hostSoABlockA.get(), elementsCount);
  SoA hostSoAR(hostSoABlockR.get(), elementsCount);
  
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  fillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoAA);
  fillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoAR);
  indirectCrossProductSoA<<<
    (elementsCount - 1)/deviceProperties.warpSize + 1,
    deviceProperties.warpSize,
    0, stream
  >>>(deviceSoAR, deviceSoAA, deviceSoAA, elementsCount);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockA.get(), deviceSoABlockA.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockR.get(), deviceSoABlockR.get(), SoA::computeDataSize(hostSoAR.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoAR.nElements(); i++)
    checkSoAelementNullityRealtiveToSquare(hostSoAR, hostSoAA, i, std::numeric_limits<double>::epsilon(), pass);
  if (!pass) {
    std::cout << "In " << typeid(*this).name() << " check failed at i= " << i << ")" << std::endl;
    std::cout << "result[" << i << "].x=" << hostSoAR[i].x << " .y=" << hostSoAR[i].y << " .z=" << hostSoAR[i].z << std::endl;
  } else {
    std::cout << std::endl;
    for (size_t j=0; j<10 ; ++j) {
      std::cout << "result[" << j << "]].x=" << hostSoAR[j].x << " .y=" << hostSoAR[j].y << " .z=" << hostSoAR[j].z << std::endl;
      std::cout << "A[" << j << "]].x=" << hostSoAA[j].x << " .y=" << hostSoAA[j].y << " .z=" << hostSoAA[j].z << std::endl;
    }
  }
  CPPUNIT_ASSERT(pass);
}

void testSoA::randomCrossProduct() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t streamA, streamB, streamR;
  CUDA_UNIT_CHECK(hipStreamCreate(&streamA));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamB));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamR));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlockA = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockB = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockR = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockA = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockB = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockR = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoAA(deviceSoABlockA.get(), elementsCount);
  SoA deviceSoAB(deviceSoABlockB.get(), elementsCount);
  SoA deviceSoAR(deviceSoABlockR.get(), elementsCount);
  SoA hostSoAA(hostSoABlockA.get(), elementsCount);
  SoA hostSoAB(hostSoABlockB.get(), elementsCount);
  SoA hostSoAR(hostSoABlockR.get(), elementsCount);
  
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamA>>>(deviceSoAA, 0xdeadbeef);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamB>>>(deviceSoAB, 0xcafefade);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamR>>>(deviceSoAR, 0xfadedcab);
  hipEvent_t eventA, eventB;
  CUDA_UNIT_CHECK(hipEventCreate(&eventA));
  CUDA_UNIT_CHECK(hipEventCreate(&eventB));
  CUDA_UNIT_CHECK(hipEventRecord(eventA, streamA));
  CUDA_UNIT_CHECK(hipEventRecord(eventB, streamB));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventA));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventB));
  indirectCrossProductSoA<<<
    (elementsCount - 1)/deviceProperties.warpSize + 1,
    deviceProperties.warpSize,
    0, streamR
  >>>(deviceSoAR, deviceSoAA, deviceSoAB, elementsCount);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockA.get(), deviceSoABlockA.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamA));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockB.get(), deviceSoABlockB.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamB));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockR.get(), deviceSoABlockR.get(), SoA::computeDataSize(hostSoAR.nElements()), hipMemcpyDeviceToHost, streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamA));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamB));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoAR.nElements(); i++) {
    checkCrossProduct(hostSoAR, hostSoAA, hostSoAB, i, std::numeric_limits<double>::epsilon(), pass);
  }
  if (!pass) {
    // Recompute the expected result
    testSoA::AoSelement expected;
    ::crossProduct(expected, hostSoAA[i], hostSoAB[i]);
    std::cout << "In " << __FUNCTION__ << " check failed at i= " << i << std::endl;
    std::cout << "result= ("   << hostSoAR[i].x << ", " << hostSoAR[i].y << ", " << hostSoAR[i].z << ")" << std::endl;
    std::cout << "expected= (" << expected.x << ", " << expected.y << ", " << expected.z << ")" << std::endl;
    std::cout << "A= (" << hostSoAA[i].x << ", " << hostSoAA[i].y << ", " << hostSoAA[i].z << ")" << std::endl;
    std::cout << "B= (" << hostSoAB[i].x << ", " << hostSoAB[i].y << ", " << hostSoAB[i].z << ")" << std::endl;
  } else {
    std::cout << std::endl;
    for (size_t j=0; j<10 && j<hostSoAR.nElements(); ++j) {
      testSoA::AoSelement expected;
      // Mixed computation AoS(single row) / SoA / SoA 
      ::crossProduct(expected, hostSoAA[j], hostSoAB[j]);
      std::cout << "In " << __FUNCTION__ << " check was OK. Sampling j= " << j << std::endl;
      std::cout << "result= ("   << hostSoAR[j].x << ", " << hostSoAR[j].y << ", " << hostSoAR[j].z << ")" << std::endl;
      std::cout << "expected= (" << expected.x << ", " << expected.y << ", " << expected.z << ")" << std::endl;
      std::cout << "difference= (" << expected.x - hostSoAR[j].x << ", " << expected.y - hostSoAR[j].y  
              << ", " << expected.z - hostSoAR[j].z << ")" << std::endl;
      std::cout << "A= (" << hostSoAA[j].x << ", " << hostSoAA[j].y << ", " << hostSoAA[j].z << ")" << std::endl;
      std::cout << "B= (" << hostSoAB[j].x << ", " << hostSoAB[j].y << ", " << hostSoAB[j].z << ")" << std::endl;
    }
  }
  CPPUNIT_ASSERT(pass);
}

void testSoA::randomCrossProductEigen() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t streamA, streamB, streamR;
  CUDA_UNIT_CHECK(hipStreamCreate(&streamA));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamB));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamR));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlockA = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockB = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockR = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockA = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockB = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockR = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoAA(deviceSoABlockA.get(), elementsCount);
  SoA deviceSoAB(deviceSoABlockB.get(), elementsCount);
  SoA deviceSoAR(deviceSoABlockR.get(), elementsCount);
  SoA hostSoAA(hostSoABlockA.get(), elementsCount);
  SoA hostSoAB(hostSoABlockB.get(), elementsCount);
  SoA hostSoAR(hostSoABlockR.get(), elementsCount);
  
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamA>>>(deviceSoAA, 0xdeadbeef);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamB>>>(deviceSoAB, 0xcafefade);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamR>>>(deviceSoAR, 0xfadedcab);
  hipEvent_t eventA, eventB;
  CUDA_UNIT_CHECK(hipEventCreate(&eventA));
  CUDA_UNIT_CHECK(hipEventCreate(&eventB));
  CUDA_UNIT_CHECK(hipEventRecord(eventA, streamA));
  CUDA_UNIT_CHECK(hipEventRecord(eventB, streamB));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventA));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventB));
  const size_t stride = (((elementsCount * sizeof(double) - 1) / deviceSoAA.byteAlignment() ) + 1) * deviceSoAA.byteAlignment() / sizeof(double);
  eigenCrossProductSoA<<<
    (elementsCount - 1)/deviceProperties.warpSize + 1,
    deviceProperties.warpSize,
    0, streamR
  >>>(deviceSoAR.x(), deviceSoAA.x(), deviceSoAB.x(), elementsCount, stride);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockA.get(), deviceSoABlockA.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamA));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockB.get(), deviceSoABlockB.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamB));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockR.get(), deviceSoABlockR.get(), SoA::computeDataSize(hostSoAR.nElements()), hipMemcpyDeviceToHost, streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamA));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamB));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoAR.nElements(); i++) {
    checkCrossProduct(hostSoAR, hostSoAA, hostSoAB, i, std::numeric_limits<double>::epsilon(), pass);
  }
  if (!pass) {
    // Recompute the expected result
    testSoA::AoSelement expected;
    ::crossProduct(expected, hostSoAA[i], hostSoAB[i]);
    std::cout << "In " << __FUNCTION__ << " check failed at i= " << i << std::endl;
    std::cout << "result= ("   << hostSoAR[i].x << ", " << hostSoAR[i].y << ", " << hostSoAR[i].z << ")" << std::endl;
    std::cout << "expected= (" << expected.x << ", " << expected.y << ", " << expected.z << ")" << std::endl;
    std::cout << "A= (" << hostSoAA[i].x << ", " << hostSoAA[i].y << ", " << hostSoAA[i].z << ")" << std::endl;
    std::cout << "B= (" << hostSoAB[i].x << ", " << hostSoAB[i].y << ", " << hostSoAB[i].z << ")" << std::endl;
  } else {
    std::cout << std::endl;
    for (size_t j=0; j<10 && j<hostSoAR.nElements(); ++j) {
      testSoA::AoSelement expected;
      // Mixed computation AoS(single row) / SoA / SoA 
      ::crossProduct(expected, hostSoAA[j], hostSoAB[j]);
      std::cout << "In " << __FUNCTION__ << " check was OK. Sampling j= " << j << std::endl;
      std::cout << "result= ("   << hostSoAR[j].x << ", " << hostSoAR[j].y << ", " << hostSoAR[j].z << ")" << std::endl;
      std::cout << "expected= (" << expected.x << ", " << expected.y << ", " << expected.z << ")" << std::endl;
      std::cout << "difference= (" << expected.x - hostSoAR[j].x << ", " << expected.y - hostSoAR[j].y  
              << ", " << expected.z - hostSoAR[j].z << ")" << std::endl;
      std::cout << "A= (" << hostSoAA[j].x << ", " << hostSoAA[j].y << ", " << hostSoAA[j].z << ")" << std::endl;
      std::cout << "B= (" << hostSoAB[j].x << ", " << hostSoAB[j].y << ", " << hostSoAB[j].z << ")" << std::endl;
    }
  }
  CPPUNIT_ASSERT(pass);
}