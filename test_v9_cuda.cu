#include "hip/hip_runtime.h"
#include <limits>

#include "soa_v9_cuda.h"

#include <hiprand/hiprand_kernel.h>

#include <Eigen/Core>
#include <Eigen/Geometry>

#define CUDA_UNIT_CHECK(A) CPPUNIT_ASSERT_EQUAL(hipSuccess, A)

namespace {
  // fill element
  template <class T>
  __host__ __device__ __forceinline__ void fillElement(T & e, size_t i) {
    e.x = 11.0 * i;
    e.y = 22.0 * i;
    e.z = 33.0 * i;
    e.colour = i;
    e.value = 0x10001 * i;
    e.py = &e.y;
  }

  // Fill up the elements of the SoA
  [[maybe_unused]] __global__ void fillSoA(testSoA::SoA soa) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= soa.nElements()) return;
    // compiler does not belive we can use a temporary soa[i] to store results.
    // So make an lvalue.
    auto e = soa[i];
    fillElement(e, i);
  }
  // A read only decorator adding x,y,z interface to Eigen vectors
  /*template <class C>
  class EigenXYZW {
  public:
    EigenXYZW(const C &v): x(v, &C::x()), y(v, &C::y()), z(v, &C::z()), w(v, &C::w())  {}
    class Accessor {
      friend EigenXYZW<C>;
    public:
      operator C::Scalar() const { return (v_.*m_)(); }
    private:
      Accessor(const C& v, C::Scalar (C::Type::*m)()): v_(v), m_(m) {}
      const C & v_;
      C::Scalar (C::Type::*m_)();
    };
    Accessor x;
    Accessor y;
    Accessor z;
    Accessor w;
  };*/
  
  // Fill elements with random data.
  [[maybe_unused]] __global__ void randomFillSoA(testSoA::SoA soa, uint64_t seed) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= soa.nElements()) return;
    hiprandState state;
    hiprand_init(seed + i, 0, 0, &state);
    soa[i].x = hiprand_uniform_double(&state);
    soa[i].y = hiprand_uniform_double(&state);
    soa[i].z = hiprand_uniform_double(&state);
    soa[i].a()(0) = hiprand_uniform_double(&state);
    soa[i].a()(1) = hiprand_uniform_double(&state);
    soa[i].a()(2) = hiprand_uniform_double(&state);
    soa[i].b()(0) = hiprand_uniform_double(&state);
    soa[i].b()(1) = hiprand_uniform_double(&state);
    soa[i].b()(2) = hiprand_uniform_double(&state);
  }

  [[maybe_unused]] __global__ void fillAoS(testSoA::AoSelement *aos, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    fillElement(aos[i], i);
  }
  
  // Simple cross product for elements
  template <typename T, typename T2>
  [[maybe_unused]] __host__ __device__ __forceinline__ void crossProduct(T & r, const T2 & __restrict__ a, const T2 & __restrict__ b) {
    r.x = a.y * b.z - a.z * b.y;
    r.y = a.z * b.x - a.x * b.z;
    r.z = a.x * b.y - a.y * b.x;
  }

  // Simple indirect cross product (SoA)
  [[maybe_unused]] __global__ void indirectCrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    // C++ does not allow creating non-const references to temporary variables
    // this workaround makes the temporary variable 
    auto ri = r[i];
    crossProduct(ri, a[i], b[i]);
  }

  // Simple direct cross product (SoA)
  [[maybe_unused]] __global__ void directCrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    r[i].x = a[i].y * b[i].z - a[i].z * b[i].y;
    r[i].y = a[i].z * b[i].x - a[i].x * b[i].z;
    r[i].z = a[i].x * b[i].y - a[i].y * b[i].x;
  }

  // Hand-made cross product as a reference (SoA)
  [[maybe_unused]] __global__ void handcraftedCrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    r.x()[i] = a.y()[i] * b.z()[i] - a.z()[i] * b.y()[i];
    r.y()[i] = a.z()[i] * b.x()[i] - a.x()[i] * b.z()[i];
    r.z()[i] = a.x()[i] * b.y()[i] - a.y()[i] * b.x()[i];
  }
  
  using V3 = Eigen::Vector3d;
  using DynStride = Eigen::InnerStride<Eigen::Dynamic>;
  using CStride = Eigen::InnerStride<1024>;
  using MapV3 =  Eigen::Map<V3,0, DynStride>;
  using CMapV3 =  Eigen::Map<const V3,0,  DynStride>;
  
   // Eigen based cross product
  [[maybe_unused]] __global__ void eigenCrossProductSoA(double* rx, const double* __restrict__ ax, const double* __restrict__ bx, size_t nElements, size_t stride) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;

    CMapV3 ma(ax+i, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(stride));
    CMapV3 mb(bx+i, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(stride));
    MapV3 mr(rx+i, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(stride));
    mr = ma.cross(mb);
  }
  
  // Eigen based cross product on embedded vectors
  [[maybe_unused]] __global__ void embeddedCrossProductSoA(testSoA::SoA soa) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= soa.nElements()) return;
#if 0
    soa[i].r() = soa[i].a().cross(soa[i].b());
#else
    using V3 = Eigen::Vector3d;
    using DynStride = Eigen::InnerStride<Eigen::Dynamic>;
    using CMapV3 =  Eigen::Map<const V3,0,  DynStride>;
    const V3::Scalar * __restrict__ mad = soa[i].a().data();
    const V3::Scalar * __restrict__ mbd = soa[i].b().data();
    CMapV3 ma(mad, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(soa[i].a.stride()));
    CMapV3 mb(mbd, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(soa[i].b.stride()));
    soa[i].r() = ma.cross(mb);
#endif
  }

   // Eigen based cross product on embedded vectors
  [[maybe_unused]] __global__ void embeddedCrossProductLocalObjectSoA(std::byte * data, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    bool deviceConstructor = true;
    testSoA::SoA soa(deviceConstructor, data, nElements);
#if 0
    soa[i].r() = soa[i].a().cross(soa[i].b());
#else
    using V3 = Eigen::Vector3d;
    using DynStride = Eigen::InnerStride<Eigen::Dynamic>;
    using CMapV3 =  Eigen::Map<const V3,0,  DynStride>;
    const V3::Scalar * __restrict__ mad = soa[i].a().data();
    const V3::Scalar * __restrict__ mbd = soa[i].b().data();
    CMapV3 ma(mad, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(soa[i].a.stride()));
    CMapV3 mb(mbd, V3::RowsAtCompileTime, V3::ColsAtCompileTime, DynStride(soa[i].b.stride()));
    soa[i].r() = ma.cross(mb);
#endif
  }

  // Simple cross product (SoA on CPU)
  [[maybe_unused]] __host__ void indirectCPUcrossProductSoA(testSoA::SoA r, const testSoA::SoA a, const testSoA::SoA b, size_t nElements) {
    for (size_t i =0; i< nElements; ++i) {
      // This version is also affected.
      auto ri = r[i];
      crossProduct(ri, a[i], b[i]);
    }
  }

  // Simple cross product (AoS)
  [[maybe_unused]] __global__ void crossProductAoS(testSoA::AoSelement *r,
          testSoA::AoSelement *a, testSoA::AoSelement *b, size_t nElements) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nElements) return;
    crossProduct(r[i], a[i], b[i]);
  }

  [[maybe_unused]] void hexdump(void *ptr, int buflen) {
    /* From https://stackoverflow.com/a/29865 with adaptations */
    unsigned char *buf = (unsigned char*)ptr;
    int i, j;
    for (i=0; i<buflen; i+=16) {
      printf("%06x: ", i);
      for (j=0; j<16; j++) {
        if (i+j < buflen)
          printf("%02x ", buf[i+j]);
        else
          printf("   ");
        if ((i+j) % 4 == 3) printf (" ");
      }
      printf(" ");
  //  for (j=0; j<16; j++)
  //    if (i+j < buflen)
  //      printf("%c", isprint(buf[i+j]) ? buf[i+j] : '.');
      printf("\n");
    }
  }
  // Check we find what we wanted to initialize.
  // Pass should be initialized to true.
  template <class T>
  [[maybe_unused]] __host__ __device__ __forceinline__ void checkSoAelement(T soa, size_t i, bool & pass) {
    if (i >= soa.nElements() || !pass) return;
    if (soa[i].x != 11.0 * i) { pass = false; return; }
    if (soa[i].y != 22.0 * i) { pass = false; return; }
    if (soa[i].z != 33.0 * i) { pass = false; return; }
    if (soa[i].colour != i) { pass = false; return; }
    if (soa[i].value != static_cast<int32_t>(0x10001 * i)) { pass = false; return; }
  }

  // Check r[i].{x, y, z} are close enough to zero compared to a[i].{x,y,z} and b[i].{x,y,z}
  // to validate a cross product of a vector with itself produced a zero (enough) result.
  template <class T>
  [[maybe_unused]] __host__ __device__ __forceinline__ void checkSoAelementNullityRealtiveToSquare(T resSoA, T referenceSoA, size_t i, double epsilon, bool & pass) {
    if (i >= resSoA.nElements() || !pass) return;
    auto ref = max (abs(referenceSoA[i].x), 
                    max(abs(referenceSoA[i].y), 
                        abs(referenceSoA[i].z)));
    ref *= ref * epsilon;
    if (abs(resSoA[i].x) > ref) { pass = false; return; }
    if (abs(resSoA[i].y) > ref) { pass = false; return; }
    if (abs(resSoA[i].z) > ref) { pass = false; return; }    
  }

  // Check r[i].{x, y, z} are close enough to zero compared to a[i].{x,y,z} and b[i].{x,y,z}
  // to validate a cross product of a vector with itself produced a zero (enough) result.
  template <class T>
  [[maybe_unused]] __host__ __device__ __forceinline__ void checkCrossProduct(T resultSoA, T aSoA, T bSoA, size_t i, double epsilon, bool & pass) {
    if (i >= resultSoA.nElements() || !pass) return;
    auto refA = max (abs(aSoA[i].x), 
                     max(abs(aSoA[i].y), 
                         abs(aSoA[i].z)));
    auto refB = max (abs(bSoA[i].x), 
                     max(abs(bSoA[i].y), 
                         abs(bSoA[i].z)));
    auto ref = max(refA, refB);
    ref *= ref * epsilon;
    testSoA::AoSelement myRes;
    crossProduct(myRes, aSoA[i], bSoA[i]);
    if (abs(myRes.x - resultSoA[i].x) > ref) { pass = false; return; }
    if (abs(myRes.y - resultSoA[i].y) > ref) { pass = false; return; }
    if (abs(myRes.z - resultSoA[i].z) > ref) { pass = false; return; }
  }
  
  template <class T>
  [[maybe_unused]] __host__ __device__ __forceinline__ void checkEmbeddedCrossProduct(T soa, size_t i, double epsilon, bool & pass) {
    if (i >= soa.nElements() || !pass) return;
    auto & a = soa[i].a();
    auto & b = soa[i].b();
    auto & r = soa[i].r();
    auto refA = max (abs(a.x()), max(abs(a.y()), abs(a.z())));
    auto refB = max (abs(b.x()), max(abs(b.y()), abs(b.z())));
    auto ref = max(refA, refB);
    ref *= ref * epsilon;
    testSoA::AoSelement myRes, AoSa, AoSb;
    AoSa.x = a.x();
    AoSa.y = a.y();
    AoSa.z = a.z();
    AoSb.x = b.x();
    AoSb.y = b.y();
    AoSb.z = b.z();
    crossProduct(myRes, AoSa, AoSb);
    if (abs(myRes.x - r.x()) > ref) { pass = false; return; }
    if (abs(myRes.y - r.y()) > ref) { pass = false; return; }
    if (abs(myRes.z - r.z()) > ref) { pass = false; return; }
  }

  class StreamTimer {
  public:
    StreamTimer() {
      if (hipSuccess != hipEventCreate(&beginning_) 
          || hipSuccess != hipEventCreate(&end_))
        throw std::bad_alloc();
    }
    
    ~StreamTimer() {
      hipEventDestroy(beginning_);
      hipEventDestroy(end_);
    }
    
    void start(hipStream_t s) {
      CUDA_UNIT_CHECK(hipEventRecord(beginning_, s));
    }
    
    void stop(hipStream_t s) {
      CUDA_UNIT_CHECK(hipEventRecord(end_, s));
      
    }
    
    float mSecs() {
      CUDA_UNIT_CHECK(hipEventSynchronize(beginning_));
      CUDA_UNIT_CHECK(hipEventSynchronize(end_));
      float ret;
      CUDA_UNIT_CHECK(hipEventElapsedTime(&ret, beginning_, end_));
      return ret;
    }
    
  private:
    hipEvent_t beginning_, end_;
  };
} // Anonymous namesapce

void testSoA::fill() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  CUDA_UNIT_CHECK(hipSetDevice(defaultDevice));  
  hipStream_t stream;
  auto e = hipStreamCreate(&stream);
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors
  auto deviceSoABlock = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlock = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoA(deviceSoABlock.get(), elementsCount);
  SoA hostSoA(hostSoABlock.get(), elementsCount);
  
  // Call kernel, get result
  fillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoA);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlock.get(), deviceSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoA.nElements(); i++) checkSoAelement(hostSoA, i, pass);
  if (!pass) {
    std::cout << "In " << typeid(*this).name() << " check failed at i= " << i << ")" << std::endl;
    hexdump(hostSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()));
    printf("base=%p, &y=%p\n", deviceSoABlock.get(), deviceSoA.y());
  }
  CPPUNIT_ASSERT(pass);
}

void testSoA::randomFill() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  CUDA_UNIT_CHECK(hipSetDevice(defaultDevice));  
  hipStream_t stream;
  auto e = hipStreamCreate(&stream);
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors
  auto deviceSoABlock = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlock = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoA(deviceSoABlock.get(), elementsCount);
  SoA hostSoA(hostSoABlock.get(), elementsCount);
  
  // Call kernel, get result
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoA, 0xbaddeed5);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlock.get(), deviceSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));
}

void testSoA::crossProduct() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t stream;
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlockA = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockR = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockA = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockR = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoAA(deviceSoABlockA.get(), elementsCount);
  SoA deviceSoAR(deviceSoABlockR.get(), elementsCount);
  SoA hostSoAA(hostSoABlockA.get(), elementsCount);
  SoA hostSoAR(hostSoABlockR.get(), elementsCount);
  
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  fillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoAA);
  fillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoAR);
  indirectCrossProductSoA<<<
    (elementsCount - 1)/deviceProperties.warpSize + 1,
    deviceProperties.warpSize,
    0, stream
  >>>(deviceSoAR, deviceSoAA, deviceSoAA, elementsCount);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockA.get(), deviceSoABlockA.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockR.get(), deviceSoABlockR.get(), SoA::computeDataSize(hostSoAR.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoAR.nElements(); i++)
    checkSoAelementNullityRealtiveToSquare(hostSoAR, hostSoAA, i, std::numeric_limits<double>::epsilon(), pass);
  if (!pass) {
    std::cout << "In " << typeid(*this).name() << " check failed at i= " << i << ")" << std::endl;
    std::cout << "result[" << i << "].x=" << hostSoAR[i].x << " .y=" << hostSoAR[i].y << " .z=" << hostSoAR[i].z << std::endl;
  }
  CPPUNIT_ASSERT(pass);
}

void testSoA::randomCrossProduct() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t streamA, streamB, streamR;
  CUDA_UNIT_CHECK(hipStreamCreate(&streamA));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamB));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamR));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlockA = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockB = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockR = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockA = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockB = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockR = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoAA(deviceSoABlockA.get(), elementsCount);
  SoA deviceSoAB(deviceSoABlockB.get(), elementsCount);
  SoA deviceSoAR(deviceSoABlockR.get(), elementsCount);
  SoA hostSoAA(hostSoABlockA.get(), elementsCount);
  SoA hostSoAB(hostSoABlockB.get(), elementsCount);
  SoA hostSoAR(hostSoABlockR.get(), elementsCount);
  
  // Timer to measure performance
  ::StreamTimer timer;
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamA>>>(deviceSoAA, 0xdeadbeef);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamB>>>(deviceSoAB, 0xcafefade);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamR>>>(deviceSoAR, 0xfadedcab);
  hipEvent_t eventA, eventB;
  CUDA_UNIT_CHECK(hipEventCreate(&eventA));
  CUDA_UNIT_CHECK(hipEventCreate(&eventB));
  CUDA_UNIT_CHECK(hipEventRecord(eventA, streamA));
  CUDA_UNIT_CHECK(hipEventRecord(eventB, streamB));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventA));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventB));
  // Run more to gather statistics
  timer.start(streamR);
  for (size_t i=0; i<20; ++i) {
    indirectCrossProductSoA<<<
      (elementsCount - 1)/deviceProperties.warpSize + 1,
      deviceProperties.warpSize,
      0, streamR
    >>>(deviceSoAR, deviceSoAA, deviceSoAB, elementsCount);
  }
  timer.stop(streamR);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockA.get(), deviceSoABlockA.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamA));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockB.get(), deviceSoABlockB.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamB));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockR.get(), deviceSoABlockR.get(), SoA::computeDataSize(hostSoAR.nElements()), hipMemcpyDeviceToHost, streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamA));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamB));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoAR.nElements(); i++) {
    checkCrossProduct(hostSoAR, hostSoAA, hostSoAB, i, std::numeric_limits<double>::epsilon(), pass);
  }
  if (!pass) {
    // Recompute the expected result
    testSoA::AoSelement expected;
    ::crossProduct(expected, hostSoAA[i], hostSoAB[i]);
    std::cout << "In " << __FUNCTION__ << " check failed at i= " << i << std::endl;
    std::cout << "result= ("   << hostSoAR[i].x << ", " << hostSoAR[i].y << ", " << hostSoAR[i].z << ")" << std::endl;
    std::cout << "expected= (" << expected.x << ", " << expected.y << ", " << expected.z << ")" << std::endl;
    std::cout << "A= (" << hostSoAA[i].x << ", " << hostSoAA[i].y << ", " << hostSoAA[i].z << ")" << std::endl;
    std::cout << "B= (" << hostSoAB[i].x << ", " << hostSoAB[i].y << ", " << hostSoAB[i].z << ")" << std::endl;
  }
  std::cout << "indirectCrossProductSoA time=" << timer.mSecs() * 1000 << " us." << std::endl;
  CPPUNIT_ASSERT(pass);
}


void testSoA::randomCrossProductEmbeddedVector() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t stream;
  CUDA_UNIT_CHECK(hipStreamCreate(&stream));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlock = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlock = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoA(deviceSoABlock.get(), elementsCount);
  SoA hostSoA(hostSoABlock.get(), elementsCount);
 
  
  // Timer to measure performance
  ::StreamTimer timer, timer2;
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, stream>>>(deviceSoA, 0xdeadbeef);
  // Run more to gather statistics
  timer.start(stream);
  for (size_t i=0; i<20; ++i) {
    embeddedCrossProductLocalObjectSoA<<<
      (elementsCount - 1)/deviceProperties.warpSize + 1,
      deviceProperties.warpSize,
      0, stream
    >>>(deviceSoABlock.get(), elementsCount);
  }
  timer.stop(stream);
  timer2.start(stream);
  for (size_t i=0; i<20; ++i) {
    embeddedCrossProductSoA<<<
      (elementsCount - 1)/deviceProperties.warpSize + 1,
      deviceProperties.warpSize,
      0, stream
    >>>(deviceSoA);
  }
  timer2.stop(stream);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlock.get(), deviceSoABlock.get(), SoA::computeDataSize(hostSoA.nElements()), hipMemcpyDeviceToHost, stream));
  CUDA_UNIT_CHECK(hipStreamSynchronize(stream));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoA.nElements(); i++) {
    checkEmbeddedCrossProduct(hostSoA, i, std::numeric_limits<double>::epsilon(), pass);
  }
  if (!pass) {
    // Recompute the expected result
    Eigen::Vector3d expected;
    auto a = hostSoA[i].a();
    auto b = hostSoA[i].b();
    auto r = hostSoA[i].r();
    expected =  a.cross(b);
    std::cout << "In " << __FUNCTION__ << " check failed at i= " << i << std::endl;
    std::cout << "result= ("   << r.x() << ", " << r.y() << ", " << r.z() << ")" << std::endl;
    std::cout << "expected= (" << expected.x() << ", " << expected.y() << ", " << expected.z() << ")" << std::endl;
    std::cout << "A= (" << a.x() << ", " << a.y() << ", " << a.z() << ")" << std::endl;
    std::cout << "B= (" << b.x() << ", " << b.y() << ", " << b.z() << ")" << std::endl;
  }
  std::cout << "embeddedCrossProductLocalObjectSoA time=" << timer.mSecs() * 1000 << " us." << std::endl;
  std::cout << "embeddedCrossProductSoA time=" << timer2.mSecs() * 1000 << " us." << std::endl;
  CPPUNIT_ASSERT(pass);
}

void testSoA::randomCrossProductEigen() {
  // Get device, stream, memory
  hipDeviceProp_t deviceProperties;
  int deviceCount=0;
  CUDA_UNIT_CHECK(hipGetDeviceCount(&deviceCount));
  CPPUNIT_ASSERT(deviceCount > 0);
  CUDA_UNIT_CHECK(hipGetDeviceProperties(&deviceProperties, defaultDevice));
  hipStream_t streamA, streamB, streamR;
  CUDA_UNIT_CHECK(hipStreamCreate(&streamA));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamB));
  CUDA_UNIT_CHECK(hipStreamCreate(&streamR));
  
  // Allocate memory and populate SoA descriptors (device A as source and R as result of cross product)
  auto deviceSoABlockA = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockB = make_device_unique(SoA::computeDataSize(elementsCount));
  auto deviceSoABlockR = make_device_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockA = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockB = make_host_unique(SoA::computeDataSize(elementsCount));
  auto hostSoABlockR = make_host_unique(SoA::computeDataSize(elementsCount));
  SoA deviceSoAA(deviceSoABlockA.get(), elementsCount);
  SoA deviceSoAB(deviceSoABlockB.get(), elementsCount);
  SoA deviceSoAR(deviceSoABlockR.get(), elementsCount);
  SoA hostSoAA(hostSoABlockA.get(), elementsCount);
  SoA hostSoAB(hostSoABlockB.get(), elementsCount);
  SoA hostSoAR(hostSoABlockR.get(), elementsCount);
  
  // Timer to measure performance
  ::StreamTimer timer;
  // Call kernels, get result. Also fill up result SoA to ensure the results go in the right place.
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamA>>>(deviceSoAA, 0xdeadbeef);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamB>>>(deviceSoAB, 0xcafefade);
  randomFillSoA<<<(elementsCount - 1)/deviceProperties.warpSize + 1, deviceProperties.warpSize, 0, streamR>>>(deviceSoAR, 0xfadedcab);
  hipEvent_t eventA, eventB;
  CUDA_UNIT_CHECK(hipEventCreate(&eventA));
  CUDA_UNIT_CHECK(hipEventCreate(&eventB));
  CUDA_UNIT_CHECK(hipEventRecord(eventA, streamA));
  CUDA_UNIT_CHECK(hipEventRecord(eventB, streamB));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventA));
  CUDA_UNIT_CHECK(hipStreamWaitEvent(streamR, eventB));
  const size_t stride = (((elementsCount * sizeof(double) - 1) / deviceSoAA.byteAlignment() ) + 1) * deviceSoAA.byteAlignment() / sizeof(double);
  // Run more to gather statistics
  timer.start(streamR);
  for (size_t i=0; i<20; ++i) {
    eigenCrossProductSoA<<<
      (elementsCount - 1)/deviceProperties.warpSize + 1,
      deviceProperties.warpSize,
      0, streamR
    >>>(deviceSoAR.x(), deviceSoAA.x(), deviceSoAB.x(), elementsCount, stride);
  }
  timer.stop(streamR);
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockA.get(), deviceSoABlockA.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamA));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockB.get(), deviceSoABlockB.get(), SoA::computeDataSize(hostSoAA.nElements()), hipMemcpyDeviceToHost, streamB));
  CUDA_UNIT_CHECK(hipMemcpyAsync(hostSoABlockR.get(), deviceSoABlockR.get(), SoA::computeDataSize(hostSoAR.nElements()), hipMemcpyDeviceToHost, streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamR));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamA));
  CUDA_UNIT_CHECK(hipStreamSynchronize(streamB));

  // Validate result
  bool pass = true;
  size_t i = 0;
  for (; pass && i< hostSoAR.nElements(); i++) {
    checkCrossProduct(hostSoAR, hostSoAA, hostSoAB, i, std::numeric_limits<double>::epsilon(), pass);
  }
  if (!pass) {
    // Recompute the expected result
    testSoA::AoSelement expected;
    ::crossProduct(expected, hostSoAA[i], hostSoAB[i]);
    std::cout << "In " << __FUNCTION__ << " check failed at i= " << i << std::endl;
    std::cout << "result= ("   << hostSoAR[i].x << ", " << hostSoAR[i].y << ", " << hostSoAR[i].z << ")" << std::endl;
    std::cout << "expected= (" << expected.x << ", " << expected.y << ", " << expected.z << ")" << std::endl;
    std::cout << "A= (" << hostSoAA[i].x << ", " << hostSoAA[i].y << ", " << hostSoAA[i].z << ")" << std::endl;
    std::cout << "B= (" << hostSoAB[i].x << ", " << hostSoAB[i].y << ", " << hostSoAB[i].z << ")" << std::endl;
  }
  std::cout << "eigenCrossProductSoA time=" << timer.mSecs() * 1000 << " us." << std::endl;
  CPPUNIT_ASSERT(pass);
}
